#include "hip/hip_runtime.h"


#include <meterpu.h>

	__global__ 
void hello(char *a, int *b) 
{
	a[threadIdx.x] += b[threadIdx.x];
}


inline void hello_world()
{
	// This is the REAL "hello world" for CUDA!
	// It takes the string "Hello ", prints it, then passes it to CUDA with an array
	// of offsets. Then the offsets are added in parallel to produce the string "World!"
	// By Ingemar Ragnemalm 2010

	using namespace meterpu;

	meter<CUDA_Time> gpu_meter;


	const int N = 16; 
	const int blocksize = 16; 


	char a[N] = "Hello \0\0\0\0\0\0";
	int b[N] = {15, 10, 6, 0, -11, 1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0};

	char *ad;
	int *bd;
	const long unsigned int csize = N*sizeof(char);
	const long unsigned int isize = N*sizeof(int);

	/*printf("%s", a);*/

	hipMalloc( (void**)&ad, csize ); 
	hipMalloc( (void**)&bd, isize ); 
	hipMemcpy( ad, a, csize, hipMemcpyHostToDevice ); 
	hipMemcpy( bd, b, isize, hipMemcpyHostToDevice ); 

	dim3 dimBlock( blocksize, 1 );
	dim3 dimGrid( 1, 1 );
	gpu_meter.start();
	hello<<<dimGrid, dimBlock>>>(ad, bd);
	hipDeviceSynchronize();
	gpu_meter.stop();
	gpu_meter.calc();
	gpu_meter.show_meter_reading();
	hipMemcpy( a, ad, csize, hipMemcpyDeviceToHost ); 
	hipFree( ad );
	hipFree( bd );

	/*printf("%s\n", a);*/

}



int main()
{


	hello_world();

	return 0;

}
