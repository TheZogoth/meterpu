

#include <meterpu.h>
#include <cuda_call.h>
#include <stdio.h>


const int N = 16; 
const int blocksize = 16; 

int main()
{


	char a[N] = "Hello \0\0\0\0\0\0";
	int b[N] = {15, 10, 6, 0, -11, 1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0};
	double c[N];

	char *ad;
	int *bd;
	double *cd;
	const int csize = N*sizeof(char);
	const int isize = N*sizeof(int);
	const int dsize = N*sizeof(double);

	printf("Before component call: %s\n", a);

	hipMalloc( (void**)&ad, csize ); 
	hipMalloc( (void**)&bd, isize ); 
	hipMalloc( (void**)&cd, dsize ); 
	hipMemcpy( ad, a, csize, hipMemcpyHostToDevice ); 
	hipMemcpy( bd, b, isize, hipMemcpyHostToDevice ); 
	hipMemcpy( cd, c, isize, hipMemcpyHostToDevice ); 

	dim3 dimBlock( blocksize, 1 );
	dim3 dimGrid( 1, 1 );

	{
		using namespace meterpu;

		//Initialize a meter with GPU energy of default device id 0
		meter< NVML_Energy<> > meter0;
		//Initialize a meter with GPU energy of device id 1
		/*Meter< NVML_Energy<1> > em;*/


		meter0.start();

		//Do sth here
		hello<<<dimGrid, dimBlock>>>(ad, bd, cd);
		hipDeviceSynchronize();

		meter0.stop();

		meter0.calc();

		hipMemcpy( a, ad, csize, hipMemcpyDeviceToHost ); 
		hipMemcpy( c, cd, dsize, hipMemcpyDeviceToHost ); 
		hipFree( ad );
		hipFree( bd );

		printf("After component call: %s\n", a);

		/*std::cout<<"Energy consumed is: "<<meter.get_value()<<" milliJ."<<std::endl;*/
		meter0.show_meter_reading();



		/* Try again */
		std::cout<<"Try a sleep kernel..."<<std::endl;
		meter+.start();

		//Do sth here
		sleep(1);

		meter0.stop();

		meter0.calc();

		/*std::cout<<"Energy consumed is: "<<meter.get_value()<<" milliJ."<<std::endl;*/
		meter0.show_meter_reading();


	}

}
